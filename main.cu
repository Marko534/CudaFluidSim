#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <glm/glm.hpp>

hipError_t ercall;
#define CCALL(call)                                                                                                    \
    ercall = call;                                                                                                     \
    if (hipSuccess != ercall)                                                                                         \
    {                                                                                                                  \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(ercall)); \
        exit(EXIT_FAILURE);                                                                                            \
    }

#define grid_l 480
#define grid_h 270

#define overrelax_const 1.0f

__device__ char *vectors;
__device__ char *vectorBuffer;

__device__ bool barrier[grid_l * grid_h];

// Const for numb of vectors
#define numHorizontal ((grid_l + 1) * grid_h)
#define numVertical ((grid_h + 1) * grid_l)

/*
 Cast the raw memory to glm::vec2 pointers
*/
#define horizontalVectors ((glm::vec2 *)vectors)
#define verticalVectors ((glm::vec2 *)(vectors + numHorizontal * sizeof(glm::vec2)))

#define horizontalVectorsBuffer ((glm::vec2 *)vectorBuffer)
#define verticalVectorsBuffer ((glm::vec2 *)(vectorBuffer + numHorizontal * sizeof(glm::vec2)))

#define horizontalVectorsCPU ((glm::vec2 *)cpuVecs)
#define verticalVectorsCPU ((glm::vec2 *)(cpuVecs + numHorizontal * sizeof(glm::vec2)))

// indexing macros remain the same
#define rightVecIndex(cellX, cellY) horizontalVecIndex(cellX + 1, cellY)
#define leftVecIndex(cellX, cellY) horizontalVecIndex(cellX, cellY)
#define upVecIndex(cellX, cellY) verticalVecIndex(cellX, cellY)
#define downVecIndex(cellX, cellY) verticalVecIndex(cellX, (cellY + 1))

#define verticalVecIndex(x, y) (x + y * (grid_l))
#define horizontalVecIndex(x, y) (x + y * (grid_l + 1))

#define inVerticalBounds(x, y) ((x) >= 0 && (x) < grid_l && (y) >= 0 && (y) <= grid_h)
#define inHorizontalBounds(x, y) ((x) >= 0 && (x) <= grid_l && (y) >= 0 && (y) < grid_h)
#define inCellBounds(x, y) ((x) >= 0 && (x) < grid_l && (y) >= 0 && (y) < grid_h)

#define cellXFromPos(p) (int)p.x
#define cellYFromPos(p) (int)p.y

inline __device__ void init_vec()
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    const int x = id % grid_l;
    const int y = id / grid_l;

    horizontalVectors[rightVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    verticalVectors[upVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    if (y == grid_h - 1)
    {
        verticalVectors[downVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    }
    if (x == 0)
    {
        horizontalVectors[leftVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    }
}

inline __device__ void init_vecBuffer()
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    const int x = id % grid_l;
    const int y = id / grid_l;

    horizontalVectorsBuffer[rightVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    verticalVectorsBuffer[upVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    if (y == grid_h - 1)
    {
        verticalVectorsBuffer[downVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    }
    if (x == 0)
    {
        horizontalVectorsBuffer[leftVecIndex(x, y)] = glm::vec2(0.0f, 0.0f);
    }
}

inline __device__ void set_horizontal_vec_cell(const glm::vec2 v, const int x, const int y)
{
    horizontalVectors[rightVecIndex(x, y)] = v;
    horizontalVectors[leftVecIndex(x, y)] = v;
}

inline __device__ void set_vertical_vec_cell(const glm::vec2 v, const int x, const int y)
{
    verticalVectors[upVecIndex(x, y)] = v;
    verticalVectors[downVecIndex(x, y)] = v;
}

__global__ void setHorizontalVec(const glm::vec2 v, const int x, const int y)
{
    set_horizontal_vec_cell(v, x, y);
}

__global__ void setVerticalVec(const glm::vec2 v, const int x, const int y)
{
    set_vertical_vec_cell(v, x, y);
}

void setHorVecs(const glm::vec2 v, const int x, const int y)
{
    setHorizontalVec<<<1, 1>>>(v, x, y);
}

void setVertVecs(const glm::vec2 v, const int x, const int y)
{
    setVerticalVec<<<1, 1>>>(v, x, y);
}

int main()
{

    std::cout << "Hello, World!" << std::endl;
    return 0;
}
