#include "hip/hip_runtime.h"
#include <GL/glew.h> // Must be included first
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hipfft/hipfft.h>

const int W = 512, H = 512;
const float dt = 0.1f, visc = 0.001f;

// CUDA-OpenGL interoperability
struct hipGraphicsResource *cuda_vbo;
GLuint vbo;

// Fluid simulation resources
hipfftHandle planR2C, planC2R;
hipArray *velArray;
hipTextureObject_t texVel; // Use hipTextureObject_t instead of texture<>

__global__ void addForces(float2 *vel, int2 pos, float2 force, int r)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H)
        return;

    int dx = x - pos.x;
    int dy = y - pos.y;
    float s = 1.f / (1.f + dx * dx * dx * dx + dy * dy * dy * dy);
    vel[y * W + x].x += s * force.x;
    vel[y * W + x].y += s * force.y;
}

__global__ void advectVel(float2 *vel, hipTextureObject_t texOldVel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H)
        return;

    float2 v;
    surf2Dread(&v, texOldVel, x * sizeof(float2), y);
    float2 pos = make_float2(x - dt * v.x, y - dt * v.y);

    float2 newVel;
    surf2Dread(&newVel, texOldVel, int(pos.x) * sizeof(float2), int(pos.y));
    vel[y * W + x] = newVel;
}

__global__ void updateParticles(float2 *particles, float2 *vel)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= W * H)
        return;

    float2 p = particles[idx];
    int x = static_cast<int>(p.x * W);
    int y = static_cast<int>(p.y * H);
    x = max(0, min(x, W - 1));
    y = max(0, min(y, H - 1));

    p.x += vel[y * W + x].x * dt;
    p.y += vel[y * W + x].y * dt;

    // Boundary wrap
    p.x = fmod(p.x + 1.f, 1.f);
    p.y = fmod(p.y + 1.f, 1.f);

    particles[idx] = p;
}

void initSimulation()
{
    // Initialize CUDA-OpenGL interop
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, W * H * sizeof(float2), 0, GL_DYNAMIC_DRAW);
    hipGraphicsGLRegisterBuffer(&cuda_vbo, vbo, cudaGraphicsMapFlagsWriteDiscard);

    // Initialize velocity field
    hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();
    hipMallocArray(&velArray, &desc, W, H);

    // Create CUFFT plans
    hipfftPlan2d(&planR2C, H, W, HIPFFT_R2C);
    hipfftPlan2d(&planC2R, H, W, HIPFFT_C2R);

    // Create and bind texture object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = velArray;

    hipTextureDesc texDesc = {};
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&texVel, &resDesc, &texDesc, nullptr);
}

void stepSimulation(int2 mousePos, float2 force)
{
    static float2 *d_vel;
    static hipfftComplex *d_velC;

    // Add forces
    dim3 blocks(W / 16, H / 16);
    dim3 threads(16, 16);
    addForces<<<blocks, threads>>>(d_vel, mousePos, force, 20);

    // Advection
    advectVel<<<blocks, threads>>>(d_vel, texVel);

    // FFT-based diffusion and projection (simplified)
    hipfftExecR2C(planR2C, (hipfftReal *)d_vel, d_velC);
    // ... Diffusion and projection in frequency domain ...
    hipfftExecC2R(planC2R, d_velC, (hipfftReal *)d_vel);

    // Update particles
    float2 *d_part;
    hipGraphicsMapResources(1, &cuda_vbo);
    size_t size;
    hipGraphicsResourceGetMappedPointer((void **)&d_part, &size, cuda_vbo);
    updateParticles<<<(W * H + 255) / 256, 256>>>(d_part, d_vel);
    hipGraphicsUnmapResources(1, &cuda_vbo);
}

void render()
{
    glClear(GL_COLOR_BUFFER_BIT);
    glEnableClientState(GL_VERTEX_ARRAY);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(2, GL_FLOAT, 0, 0);
    glDrawArrays(GL_POINTS, 0, W * H);
    glDisableClientState(GL_VERTEX_ARRAY);
}

int main()
{
    glfwInit();
    GLFWwindow *window = glfwCreateWindow(W, H, "CUDA Fluid", NULL, NULL);
    glfwMakeContextCurrent(window);
    glewInit();

    initSimulation();

    while (!glfwWindowShouldClose(window))
    {
        // Get mouse input and calculate force
        // stepSimulation(mousePos, force);
        render();
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipGraphicsUnregisterResource(cuda_vbo);
    glDeleteBuffers(1, &vbo);
    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);
    hipDestroyTextureObject(texVel);
    hipFreeArray(velArray);
    glfwTerminate();
    return 0;
}
